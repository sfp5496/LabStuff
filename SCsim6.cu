#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "SC.h"
#include <time.h>
#include <string.h>
#include "myhelpers.h"
//#include "mysyslib.h"

double3* r;
double3* rhost;
double3* rchost;
double3* r1;
double3* r1host;
double3* r2;
double3* r2host;
double3* u;
double3* uhost;
double2* theta;
double2* thetahost;
double* l;
double* lhost;
double* sigma;
double* sigmahost;
double3* GU1;
double3* GU1host;
double2* GU1A;
double2* GU1Ahost;
double3* GU0;
double2* GU0A;
double3* dr;
double2* dtheta;
double3* h;
double2* hA;
double* GUI;
double3* GUII;
double2* GUIIA;
double2* UW;
double* GS;
double* GSA;
double* U;
double* p1;
double* GShost;
double* GSAhost;
double* Uhost;
double* p1host;
double* Vars; //Vars[0] == GS; Vars[1] == GSA; Vars[2] == U; Vars[3] == p1;
double* Varshost;

size_t pitch;

dim3 blocks, threads;
dim3 grid, block;

FILE *datfile;

void doOpen(const char *fname, const char *mode) {
	fprintf(stderr, "Opening %s with mode %s\n", fname, mode);
	datfile = fopen(fname, mode);
	if (!datfile) {
		perror("fopen");
		exit(2);
	}
}

void doClose() {
	if (fclose(datfile))
		perror("fclose");
}

int safeRead(void *buf, size_t size, size_t nr) {
	size_t num = fread(buf, size, nr, datfile);
	if (num < nr) {
		if (feof(datfile)) {
			fprintf(stderr, "Reached EOF\n");
			return 1;
		}
		else {
			perror("fread");
			exit(-1);
		}
	}
	return 0;
}

int doRead(void *buf, size_t size, size_t nr) {
	size_t num = fread(buf, size, nr, datfile);
	if (num < nr) {
		if (feof(datfile)) {
			fprintf(stderr, "Reached EOF, exitting...\n");
			exit(0);
		}
		else {
			perror("fread");
			exit(-1);
		}
	}
	return 0;
}

void doWrite(const void *buf, size_t size, size_t nr) {
	if (!fwrite(buf, size, nr, datfile)) {
		perror("fwrite");
		exit(-1);
	}
}


__attribute__ ((malloc)) void * xmalloc(size_t size) {
	void *p;
	p=malloc(size);
	if (!p) {
		perror("xmalloc");
		exit(-1);
	}
	return p;
}

void xmemcpy(void *out, void *in, size_t size) {
        if (!memcpy(out, in, size)) {
		perror("xmemcpy");
		exit(1);
        }
}


__device__ __host__ double2 d2add(double2 a, double2 b) {
	double2 ret;
	ret.x=a.x+b.x;
	ret.y=a.y+b.y;
	return ret;
}

__device__ __host__ double2 d2sub(double2 a, double2 b) {
	double2 ret;
	ret.x=a.x-b.x;
	ret.y=a.y-b.y;
	return ret;
}

__device__ __host__ double2 d2multscal(double2 a, double b) {
	double2 ret;
	ret.x=b*a.x;
	ret.y=b*a.y;
	return ret;
}

__device__ __host__ double2 d2multscal2(double2 a, double b) {
	double2 ret;
	ret.x=b*b*a.x;
	ret.y=b*b*a.y;
	return ret;
}

__device__ __host__ double2 d2divscal(double2 a, double b) {
	double2 ret;
	ret.x=a.x/b;
	ret.y=a.y/b;
	return ret;
}

__device__ __host__ double d2dotp(double2 a, double2 b) {
	return (a.x*b.x)+(a.y*b.y);
}

__device__ __host__ double d2mag(double2 a) {
	return sqrt((a.x)*(a.x)+(a.y)*(a.y));
}

__device__ __host__ double d2dist(double2 a, double2 b) {
	return d2mag(d2sub(a,b));
}

__device__ __host__ double2 d2unit(double2 a) {
	return d2divscal(a, d2mag(a));
}

__device__ __host__ double3 d3add(double3 a, double3 b) {
	double3 ret;
	ret.x=a.x+b.x;
	ret.y=a.y+b.y;
	ret.z=a.z+b.z;
	return ret;
}

__device__ __host__ double3 d3sub(double3 a, double3 b) {
	double3 ret;
	ret.x=a.x-b.x;
	ret.y=a.y-b.y;
	ret.z=a.z-b.z;
	return ret;
}

__device__ __host__ double3 d3multscal(double3 a, double b) {
	double3 ret;
	ret.x=b*a.x;
	ret.y=b*a.y;
	ret.z=b*a.z;
	return ret;
}

__device__ __host__ double3 d3multscal2(double3 a, double b) {
	double3 ret;
	ret.x=b*b*a.x;
	ret.y=b*b*a.y;
	ret.z=b*b*a.z;
	return ret;
}

__device__ __host__ double3 d3divscal(double3 a, double b) {
	double3 ret;
	ret.x=a.x/b;
	ret.y=a.y/b;
	ret.z=a.z/b;
	return ret;
}

__device__ __host__ double d3dotp(double3 a, double3 b) {
	return (a.x*b.x)+(a.y*b.y)+(a.z*b.z);
}

__device__ __host__ double3 d3crossp(double3 a, double3 b) {
	double3 ret;
	ret.x=(a.y*b.z)-(a.z*b.y);
	ret.y=-(a.x*b.z)+(a.z*b.x);
	ret.z=(a.x*b.y)-(a.y*b.x);
	return ret;
}

__device__ __host__ double d3mag(double3 a) {
	return sqrt(a.x*a.x+a.y*a.y+a.z*a.z);
}

__device__ __host__ double d3dist(double3 a, double3 b) {
	return d3mag(d3sub(a,b));
}

__device__ __host__ double3 d3unit(double3 a) {
	return d3divscal(a,d3mag(a));
}

//THESE FUNCTIONS ARE FOR THE SPHEROCYLINDER CODE//

__device__ __host__ double d3SCdist(double3 ri, double3 rj, double3 ui, double3 uj, double li, double lj) {
	double3 dij;
	dij=d3sub(d3sub(d3add(rj,d3multscal(uj,lj)),ri),d3multscal(ui,li));
	return d3mag(dij);
}

__device__ __host__ double L_i(double3 ri, double3 rj, double3 ui, double3 uj, double li) {
	double retn,retd;
	retn=d3dotp(ui,d3sub(rj,ri))-d3dotp(ui,uj)*d3dotp(uj,d3sub(rj,ri));
	retd=1.0-(d3dotp(ui,uj)*d3dotp(ui,uj));
	if((retn/retd)>(li/2.0)) {
		return (li/2.0);
	}
	else if((retn/retd)<(-li/2.0)) {
		return (-li/2.0);
	}
	else {
		return (retn/retd);
	}
}

__device__ __host__ double L_j(double3 ri, double3 rj, double3 ui, double3 uj, double lj) {
	double retn,retd;
	retn=d3dotp(ui,uj)*d3dotp(ui,d3sub(rj,ri))-d3dotp(uj,d3sub(rj,ri));
	retd=1.0-(d3dotp(ui,uj)*d3dotp(ui,uj));
	if((retn/retd)>(lj/2.0)) {
		return (lj/2.0);
	}
	else if((retn/retd)<(-lj/2.0)) {
		return (-lj/2.0);
	}
	else {
		return (retn/retd);
	}
}

double unitrand() {
	return (((double)rand())/((double)(RAND_MAX)));
}

void initcond(double3 *x, double2 *ang, double *len, double *diam) {
	int i;
	srand(time(NULL));
	for(i=0;i<npart;i++) {
		x[i].x=R*unitrand();
		x[i].y=2*M_PI*unitrand();
		x[i].z=H*unitrand();
		len[i]=L;
		diam[i]=SIGMA;
		ang[i].x=unitrand()*2*M_PI;
		ang[i].y=unitrand()*2*M_PI;
	}
}

__device__ __host__ void sptoca(int i, double2* theta, double3* u) {
	u[i].x=sin(theta[i].y)*cos(theta[i].x);
	u[i].y=sin(theta[i].y)*sin(theta[i].x);
	u[i].z=cos(theta[i].y);
}

__device__ __host__ void ends(int i,double3* r,double3* u,double* l,double3* r1,double3* r2) {
	r1[i]=d3add(r[i],d3multscal(u[i],0.5*l[i]));
	r2[i]=d3sub(r[i],d3multscal(u[i],0.5*l[i]));
	r1[i].x=hypot(r1[i].x,r1[i].y);
	r2[i].x=hypot(r2[i].x,r2[i].y);
}

void start() {
	hipSetDevice(1);

	if(npart<1024) {
		blocks.x=1;
		threads.x=npart;

		grid.x=npart;
		grid.y=npart;
		grid.z=1;
		block.x=1;
		block.y=1;
		block.z=1;
	}
	else {
		blocks.x=npart/512;
		threads.x=512;

		grid.x=512;
		grid.y=512;
		grid.z=1;
		block.x=npart/512;
		block.y=npart/512;
		block.z=1;
	}
	HANDLE_ERROR(hipMalloc(&r,npart*sizeof(double3)));
	HANDLE_ERROR(hipMalloc(&r1,npart*sizeof(double3)));
	HANDLE_ERROR(hipMalloc(&r2,npart*sizeof(double3)));
	HANDLE_ERROR(hipMalloc(&u,npart*sizeof(double3)));
	HANDLE_ERROR(hipMalloc(&theta,npart*sizeof(double2)));
	HANDLE_ERROR(hipMalloc(&l,npart*sizeof(double)));
	HANDLE_ERROR(hipMalloc(&sigma,npart*sizeof(double)));
	HANDLE_ERROR(hipMalloc(&GU1,npart*sizeof(double3)));
	HANDLE_ERROR(hipMalloc(&GU1A,npart*sizeof(double2)));
	HANDLE_ERROR(hipMalloc(&GU0,npart*sizeof(double3)));
	HANDLE_ERROR(hipMalloc(&GU0A,npart*sizeof(double3)));
	HANDLE_ERROR(hipMalloc(&dr,npart*sizeof(double3)));
	HANDLE_ERROR(hipMalloc(&dtheta,npart*sizeof(double2)));
	HANDLE_ERROR(hipMalloc(&h,npart*sizeof(double3)));
	HANDLE_ERROR(hipMalloc(&hA,npart*sizeof(double2)));
	HANDLE_ERROR(hipMalloc(&GUI,npart*sizeof(double)));
	HANDLE_ERROR(hipMalloc(&GUII,npart*sizeof(double3)));
	HANDLE_ERROR(hipMalloc(&GUIIA,npart*sizeof(double2)));
	HANDLE_ERROR(hipMalloc(&UW,npart*sizeof(double2)));
	HANDLE_ERROR(hipMalloc(&GS,sizeof(double)));
	HANDLE_ERROR(hipMalloc(&GSA,sizeof(double)));
	HANDLE_ERROR(hipMalloc(&U,sizeof(double)));
	HANDLE_ERROR(hipMalloc(&p1,sizeof(double)));
	rhost=(double3*)malloc(npart*sizeof(double3));
	r1host=(double3*)malloc(npart*sizeof(double3));
	r2host=(double3*)malloc(npart*sizeof(double3));
	uhost=(double3*)malloc(npart*sizeof(double3));
	thetahost=(double2*)malloc(npart*sizeof(double2));
	lhost=(double*)malloc(npart*sizeof(double));
	sigmahost=(double*)malloc(npart*sizeof(double));
	GU1host=(double3*)malloc(npart*sizeof(double3));
	GU1Ahost=(double2*)malloc(npart*sizeof(double2));
	HANDLE_ERROR(hipHostMalloc(&Varshost,5*sizeof(double)));
	HANDLE_ERROR(hipMalloc(&Vars,5*sizeof(double)));
	HANDLE_ERROR(hipHostMalloc(&GShost,sizeof(double)));
	HANDLE_ERROR(hipHostMalloc(&GSAhost,sizeof(double)));
	HANDLE_ERROR(hipHostMalloc(&Uhost,sizeof(double)));
	HANDLE_ERROR(hipHostMalloc(&p1host,sizeof(double)));
	rchost=(double3*)malloc(npart*sizeof(double3));
	initcond(rchost,thetahost,lhost,sigmahost);
	int i;
	for(i=0;i<npart;i++) {
		rhost[i].x=rchost[i].x*cos(rchost[i].y);
		rhost[i].y=rchost[i].x*sin(rchost[i].y);
		rhost[i].z=rchost[i].z;
		sptoca(i,thetahost,uhost);
	}
	HANDLE_ERROR(hipMemcpy(r,rhost,npart*sizeof(double3),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(theta,thetahost,npart*sizeof(double2),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(u,uhost,npart*sizeof(double3),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(l,lhost,npart*sizeof(double),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(sigma,sigmahost,npart*sizeof(double),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemset(dr,0.0,npart*sizeof(double3)));
	HANDLE_ERROR(hipMemset(GU1,0.0,npart*sizeof(double3)));
	HANDLE_ERROR(hipMemset(GU1A,0.0,npart*sizeof(double2)));
}

__device__ __host__ double PotEnergyI(int i, int j, double3* r, double3* u, double2* theta, double* l, double* sigma, double3* r1, double3* r2) {
	double ret=0.0;
	sptoca(i,theta,u);
	sptoca(j,theta,u);
	ends(i,r,u,l,r1,r2);
	ends(j,r,u,l,r1,r2);
	if((i!=j) && (d3dist(r[i],r[j])<(l[i]+sigma[i]+l[j]+sigma[j])/2.0)) {
		double lambda_i, lambda_j;
		lambda_i=L_i(r[i],r[j],u[i],u[j],l[i]);
		lambda_j=L_j(r[i],r[j],u[i],u[j],l[j]);
		double d;
		d=d3SCdist(r[i],r[j],u[i],u[j],lambda_i,lambda_j);
		if(d<(sigma[i]+sigma[j])/2.0) {
			ret+=0.5*((sigma[i]+sigma[j])/2.0-d)*((sigma[i]+sigma[j])/2.0-d);
		}
	}
	return ret;
}

__device__ __host__ double WallEnergy(int i, double3* r, double3* u, double2* theta,  double* l, double* sigma, double3* r1, double3* r2) {
	double ret=0.0;
	sptoca(i,theta,u);
	ends(i,r,u,l,r1,r2);
	
	if(r1[i].x>R-(sigma[i]/2.0)) {
		ret+=0.5*(r1[i].x-(R-sigma[i]/2.0))*(r1[i].x-(R-sigma[i]/2.0));
	}
	if(r2[i].x>R-(sigma[i]/2.0)) {
		ret+=0.5*(r2[i].x-(R-sigma[i]/2.0))*(r2[i].x-(R-sigma[i]/2.0));
	}
	if(r1[i].z-sigma[i]/2.0<0.0) {
		ret+=0.5*(r1[i].z-sigma[i]/2.0)*(r1[i].z-sigma[i]/2.0);
	}
	if(r1[i].z+sigma[i]/2.0>H) {
		ret+=0.5*(r1[i].z+sigma[i]/2.0-H)*(r1[i].z+sigma[i]/2.0-H);
	}
	if(r2[i].z-sigma[i]/2.0<0.0) {
		ret+=0.5*(r2[i].z-sigma[i]/2.0)*(r2[i].z-sigma[i]/2.0);
	}
	if(r2[i].z+sigma[i]/2.0>H) {
		ret+=0.5*(r2[i].z+sigma[i]/2.0-H)*(r2[i].z+sigma[i]/2.0-H);
	}
	return ret;
}

__global__ void Zero(double* GUI, double3* GUII, double2* GUIIA) {
	int tx=threadIdx.x+blockIdx.x*blockDim.x;
	*(GUI+tx)=0.0;
	((GUII+tx)->x)=0.0;
	((GUII+tx)->y)=0.0;
	((GUII+tx)->z)=0.0;
	((GUIIA+tx)->x)=0.0;
	((GUIIA+tx)->y)=0.0;
}

__global__ void GradI(double3* r, double3* u, double2* theta, double* l, double* sigma, double3* r1, double3* r2, double* GUI, double3* GUII, double2* GUIIA) {
	int tx=threadIdx.x+blockIdx.x*blockDim.x;
	int ty=threadIdx.y+blockIdx.y*blockDim.y;

	*(GUI+tx)+=PotEnergyI(tx,ty,r,u,theta,l,sigma,r1,r2);

	double dx=.0001;
	((r+tx)->x)+=dx;
	((GUII+tx)->x)+=PotEnergyI(tx,ty,r,u,theta,l,sigma,r1,r2);
	((r+tx)->x)-=dx;

	double dy=.0001;
	((r+tx)->y)+=dy;
	((GUII+tx)->y)+=PotEnergyI(tx,ty,r,u,theta,l,sigma,r1,r2);
	((r+tx)->y)-=dy;

	double dz=.0001;
	((r+tx)->z)+=dz;
	((GUII+tx)->z)+=PotEnergyI(tx,ty,r,u,theta,l,sigma,r1,r2);
	((r+tx)->z)-=dz;

	double dt=.0001;
	((theta+tx)->x)+=dt;
	((GUIIA+tx)->x)+=PotEnergyI(tx,ty,r,u,theta,l,sigma,r1,r2);
	((theta+tx)->x)-=dt;

	double dp=.0001;
	((theta+tx)->y)+=dp;
	((GUIIA+tx)->y)+=PotEnergyI(tx,ty,r,u,theta,l,sigma,r1,r2);
	((theta+tx)->y)-=dp;
}

__global__ void GradWall(double3* r, double3* u, double2* theta, double* l, double* sigma, double3* r1, double3* r2, double* GUI, double3* GUII, double2* GUIIA) {
	int tx=threadIdx.x+blockIdx.x*blockDim.x;

	*(GUI+tx)+=WallEnergy(tx,r,u,theta,l,sigma,r1,r2);

	double dx=.0001;
	((r+tx)->x)+=dx;
	((GUII+tx)->x)+=WallEnergy(tx,r,u,theta,l,sigma,r1,r2);
	((r+tx)->x)-=dx;

	double dy=.0001;
	((r+tx)->y)+=dy;
	((GUII+tx)->y)+=WallEnergy(tx,r,u,theta,l,sigma,r1,r2);
	((r+tx)->y)-=dy;

	double dz=.0001;
	((r+tx)->z)+=dz;
	((GUII+tx)->z)+=WallEnergy(tx,r,u,theta,l,sigma,r1,r2);
	((r+tx)->z)-=dz;

	double dt=.0001;
	((theta+tx)->x)+=dt;
	((GUIIA+tx)->x)+=WallEnergy(tx,r,u,theta,l,sigma,r1,r2);
	((theta+tx)->x)-=dt;

	double dp=.0001;
	((theta+tx)->y)+=dp;
	((GUIIA+tx)->y)+=WallEnergy(tx,r,u,theta,l,sigma,r1,r2);
	((theta+tx)->y)-=dp;
}

__global__ void GradII(double* GUI, double3* GUII, double2* GUIIA, double3* GU1, double2* GU1A,double* sigma) {
	int tx=threadIdx.x+blockIdx.x*blockDim.x;

	double dx=.0001;
	((GU1+tx)->x)=(((GUII+tx)->x)-(*(GUI+tx)))/dx;

	double dy=.0001;
	((GU1+tx)->y)=(((GUII+tx)->y)-(*(GUI+tx)))/dy;

	double dz=.0001;
	((GU1+tx)->z)=(((GUII+tx)->z)-(*(GUI+tx)))/dz;

	double dt=.0001;
	((GU1A+tx)->x)=(((GUIIA+tx)->x)-(*(GUI+tx)))/dt;
	
	double dp=.0001;
	((GU1A+tx)->y)=(((GUIIA+tx)->y)-(*(GUI+tx)))/dp;

	if(sigma[tx]==0.0) {
		((GU1+tx)->x)=0.0;
		((GU1+tx)->y)=0.0;
		((GU1+tx)->z)=0.0;
		((GU1A+tx)->x)=0.0;
		((GU1A+tx)->y)=0.0;
	}
}

__global__ void ConGradI(double3* dr, double3* h, double3* GU1) {
	int tx=threadIdx.x+blockIdx.x*blockDim.x;
	
	double eta=-.001;
	h[tx]=GU1[tx];
	dr[tx]=d3multscal(h[tx],eta);
}

__global__ void ConGradII(double3* GU1, double3* GU0, double3* dr, double3* h) {
	int tx=threadIdx.x+blockIdx.x*blockDim.x;

	((dr+tx)->x)=0.0;
	((dr+tx)->y)=0.0;
	((dr+tx)->z)=0.0;
	if(d3mag(GU1[tx])>0.0) {
		double gamma;
		double eta=-.001;
		gamma=d3dotp(GU1[tx],GU1[tx])/d3dotp(GU0[tx],GU0[tx]);
		if(gamma>1.0) {
			gamma=1.0;
		}
		if(gamma<-1.0) {
			gamma=-1.0;
		}
		h[tx]=d3add(GU1[tx],d3multscal(h[tx],gamma));
		dr[tx]=d3multscal(h[tx],eta);
	}
}

__global__ void ConGradAI(double2* dtheta, double2* hA, double2* GU1A) {
	int tx=threadIdx.x+blockIdx.x*blockDim.x;

	double eta=-.001;
	hA[tx]=GU1A[tx];
	dtheta[tx]=d2multscal(hA[tx],eta);
}

__global__ void ConGradAII(double2* GU1A, double2* GU0A, double2* dtheta, double2* hA) {
	int tx=threadIdx.x+blockIdx.x*blockDim.x;

	((dtheta+tx)->x)=0.0;
	((dtheta+tx)->y)=0.0;
	if(d2mag(GU1A[tx])>0.0) {
		double gamma;
		double eta=-.001;
		gamma=d2dotp(GU1A[tx],GU1A[tx])/d2dotp(GU0A[tx],GU0A[tx]);
		if(gamma>1.0) {
			gamma=1.0;
		}
		if(gamma<-1.0) {
			gamma=-1.0;
		}
		hA[tx]=d2add(GU1A[tx],d2multscal(hA[tx],gamma));
		dtheta[tx]=d2multscal(hA[tx],eta);
	}
}

__global__ void ConGradMove(double3* r, double3* dr, double2* theta, double2* dtheta, double3* GU1, double3* GU0, double2* GU1A, double2* GU0A) {
	int tx=threadIdx.x+blockIdx.x*blockDim.x;
	
	if(tx>npart) {
		return;
	}

	r[tx]=d3add(r[tx],dr[tx]);
	GU0[tx]=GU1[tx];

	if(dtheta[tx].x>1.0) {
		dtheta[tx].x=1.0;
	}
	else if(dtheta[tx].x<-1.0) {
		dtheta[tx].x=-1.0;
	}
	if(dtheta[tx].y>1.0) {
		dtheta[tx].y=1.0;
	}
	else if(dtheta[tx].y<-1.0) {
		dtheta[tx].y=-1.0;
	}
	theta[tx]=d2add(theta[tx],dtheta[tx]);
	GU0A[tx].x=GU1A[tx].x;
	GU0A[tx].y=GU1A[tx].y;

	if((theta[tx].x>2.0*M_PI) || (theta[tx].x<0.0)) {
		double t,td;
		td=theta[tx].x/(2.0*M_PI);
		t=floor(td);
		theta[tx].x=(2.0*M_PI)*(td-t);
	}
	if((theta[tx].y>2.0*M_PI) || (theta[tx].y<0.0)) {
		double t,td;
		td=theta[tx].y/(2.0*M_PI);
		t=floor(td);
		theta[tx].y=(2.0*M_PI)*(td-t);
	}
}

__global__ void PotEnergy(double3* r, double3* u, double2* theta, double* l, double* sigma, double3* r1, double3* r2, double* U) {
	int tx=threadIdx.x+blockIdx.x*blockDim.x;
	
	int ty;
	for(ty=0;ty<npart;ty++) {
		U[2]+=PotEnergyI(tx,ty,r,u,theta,l,sigma,r1,r2);
	}
	U[2]+=WallEnergy(tx,r,u,theta,l,sigma,r1,r2);
}

double contacts() {
	int i, j;
	double ret=0.0;
	for(i=0;i<npart;i++) {
		for(j=0;j<npart;j++) {
			sptoca(i,thetahost,uhost);
			ends(i,rhost,uhost,lhost,r1host,r2host);
			if((i!=j) && (d3dist(rhost[i],rhost[j])<(lhost[i]+sigmahost[i]+lhost[j]+sigmahost[j])/2.0)) {
				double lambda_i, lambda_j;
				lambda_i=L_i(rhost[i],rhost[j],uhost[i],uhost[j],lhost[i]);
				lambda_j=L_j(rhost[i],rhost[j],uhost[i],uhost[j],lhost[j]);
				double d;
				d=d3SCdist(rhost[i],rhost[j],uhost[i],uhost[j],lambda_i,lambda_j);
				if(d<(sigmahost[i]+sigmahost[j])*(.9)/2.0) {
					ret+=1.0;
				}
			}
		}
		sptoca(i,thetahost,uhost);
		ends(i,rhost,uhost,lhost,r1host,r2host);
		if(r1host[i].x>R-(sigmahost[i]/2.0)*(.95)) {
			ret+=1.0;
		}
		if(r2host[i].x>R-(sigmahost[i]/2.0)*(.95)) {
			ret+=1.0;
		}
		if(r1host[i].z-(sigmahost[i]*.95)/2.0<0.0) {
			ret+=1.0;
		}
		if(r1host[i].z+(sigmahost[i]*.95)/2.0>H) {
			ret+=1.0;
		}
		if(r2host[i].z-(sigmahost[i]*.95)/2.0<0.0) {
			ret+=1.0;
		}
		if(r2host[i].z+(sigmahost[i]*.95)/2.0>H) {
			ret+=1.0;
		}
	}
	return ret;	
}

double packfrac() {
	double Vp=0.0;
	double Vb=M_PI*R*R*H;
	int i=0;
	Vp=npart*(M_PI*(sigmahost[i]/2.0)*(sigmahost[i]/2.0)*lhost[i]+(4.0/3.0)*M_PI*(sigmahost[i]/2.0)*(sigmahost[i]/2.0)*(sigmahost[i]/2.0));
	return Vp/Vb;
}

__global__ void Pressure(double3* r, double3* u, double2* theta, double* l, double* sigma, double3* r1, double3* r2, double* p) {
	int tx=threadIdx.x+blockIdx.x*blockDim.x;

	double3 F;
	double3 P;
	F.x=0.0;
	F.y=0.0;
	F.z=0.0;
	P.x=0.0;
	P.y=0.0;
	P.z=0.0;

	sptoca(tx,theta,u);
	ends(tx,r,u,l,r1,r2);
	if(r1[tx].x>R-(sigma[tx]/2.0)) {
		F.x+=r1[tx].x-(R-sigma[tx]/2.0);
	}
	if(r2[tx].x>R-(sigma[tx]/2.0)) {
		F.x+=r2[tx].x-(R-sigma[tx]/2.0);
	}
	if(r1[tx].z-sigma[tx]/2.0<0.0) {
		F.z+=r1[tx].z-sigma[tx]/2.0;
	}
	if(r1[tx].z+sigma[tx]/2.0>H) {
		F.y+=r1[tx].z+sigma[tx]/2.0-H;
	}
	if(r2[tx].z-sigma[tx]/2.0<0.0) {
		F.z+=r2[tx].z-sigma[tx]/2.0;
	}
	if(r2[tx].z+sigma[tx]/2.0>H) {
		F.y+=r2[tx].z+sigma[tx]/2.0-H;
	}
	P.x=F.x/(2.0*M_PI*R*H);
	P.y=F.y/(M_PI*R*R);
	P.z=F.z/(M_PI*R*R);
	p[3]+=fabs(P.x)+fabs(P.y)+fabs(P.z);
}

__global__ void GradSum(double3* GU1, double* GS) {
	int tx=threadIdx.x+blockIdx.x*blockDim.x;
	
	GS[0]+=d3mag(GU1[tx]);
}

__global__ void GradSumA(double2* GU1A, double* GSA) {
	int tx=threadIdx.x+blockIdx.x*blockDim.x;

	GSA[1]+=d2mag(GU1A[tx]);
}

__global__ void ZeroVars(double* Vars) {
	int i;
	for(i=0;i<4;i++) {
		Vars[i]=0.0;
	}
}

void doAnIter() {
	//clock_t t0,t1,t2,t3,t4,t5,t6,t7,t8,t9,t10,t11,t12,t13,t14,t15;
	//t0=clock();
	Zero<<<blocks, threads>>>(GUI,GUII,GUIIA);
	//t1=clock();
	GradI<<<grid, block>>>(r,u,theta,l,sigma,r1,r2,GUI,GUII,GUIIA);
	//t2=clock();
	GradWall<<<blocks, threads>>>(r,u,theta,l,sigma,r1,r2,GUI,GUII,GUIIA);
	//t3=clock();
	GradII<<<blocks, threads>>>(GUI,GUII,GUIIA,GU1,GU1A,sigma);
	//t4=clock();
	ConGradI<<<blocks, threads>>>(dr,h,GU1);
	//t5=clock();
	ConGradAI<<<blocks, threads>>>(dtheta,hA,GU1A);
	//t6=clock();
	ConGradMove<<<blocks, threads>>>(r,dr,theta,dtheta,GU1,GU0,GU1A,GU0A);
	//t7=clock();
	ZeroVars<<<1,1>>>(Vars);
	//t8=clock();
	GradSum<<<blocks, threads>>>(GU1,Vars);
	//t9=clock();
	GradSumA<<<blocks, threads>>>(GU1A,Vars);
	//t10=clock();
	Pressure<<<threads, blocks>>>(r,u,theta,l,sigma,r1,r2,Vars);
	PotEnergy<<<threads, blocks>>>(r,u,theta,l,sigma,r1,r2,Vars);
	HANDLE_ERROR(hipMemcpy(Varshost,Vars,5*sizeof(double),hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(rhost,r,npart*sizeof(double3),hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(uhost,u,npart*sizeof(double3),hipMemcpyDeviceToHost));
	//t11=clock();
	int i;
	Varshost[4]=1.0;
	while((fabs(Varshost[0])/(npart*lhost[1])>0.000000001) && (fabs(Varshost[1])/(npart*lhost[1])>0.00000001)) {
		Zero<<<blocks, threads>>>(GUI,GUII,GUIIA);
		GradI<<<grid, block>>>(r,u,theta,l,sigma,r1,r2,GUI,GUII,GUIIA);
		GradWall<<<blocks, threads>>>(r,u,theta,l,sigma,r1,r2,GUI,GUII,GUIIA);
		GradII<<<blocks, threads>>>(GUI,GUII,GUIIA,GU1,GU1A,sigma);
		//t12=clock();
		ConGradII<<<blocks, threads>>>(GU1,GU0,dr,h);
		//t13=clock();
		ConGradAII<<<blocks,threads>>>(GU1A,GU0A,dtheta,hA);
		//t14=clock();
		ConGradMove<<<blocks,threads>>>(r,dr,theta,dtheta,GU1,GU0,GU1A,GU0A);
		ZeroVars<<<1,1>>>(Vars);
		GradSum<<<blocks, threads>>>(GU1,Vars);
		GradSumA<<<blocks, threads>>>(GU1A,Vars);
		PotEnergy<<<threads, blocks>>>(r,u,theta,l,sigma,r1,r2,Vars);
		Pressure<<<threads, blocks>>>(r,u,theta,l,sigma,r1,r2,Vars);
		Varshost[4]=Varshost[4]+1.0;
		HANDLE_ERROR(hipMemcpy(Varshost,Vars,5*sizeof(double),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(rhost,r,npart*sizeof(double3),hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(uhost,u,npart*sizeof(double3),hipMemcpyDeviceToHost));
		fprintf(stderr,"Grad Sum: %lf\n",Varshost[0]);
	}
	//t15=clock();
	double c=contacts();
	double p=packfrac();
	Uhost[0]=Varshost[2];
	p1host[0]=Varshost[3];
	fprintf(stderr,"Rod Radius: %lf\n", sigmahost[0]/2.0);
	fprintf(stderr,"Rod Length: %lf\n", lhost[0]);
	fprintf(stderr,"Packing Fraction: %lf\n",p);
	fprintf(stderr,"Average Number of Contacts: %lf\n",c/npart);
	fprintf(stderr,"Potential Energy: %lf\n",Uhost[0]);
	fprintf(stderr,"Average Overlap: %lf\n",sqrt((Uhost[0]/npart)*2.0)/(sigmahost[0]/2.0));
	fprintf(stderr,"Pressure: %lf\n",p1host[0]);
	fprintf(stderr,"GradSum: %lf\n",Varshost[0]);
	fprintf(stderr,"GradSumA: %lf\n",Varshost[1]);
	fprintf(stdout,"%lf	%lf	%lf\n",p,p1host[0],Uhost[0]);
	/*for(i=0;i<npart;i++) {
		fprintf(stdout,"%i	%lf	%lf	%lf	%lf	%lf	%lf	%lf	%lf\n",i,rhost[i].x,rhost[i].y,rhost[i].z,uhost[i].x,uhost[i].y,uhost[i].z,sigmahost[i],lhost[i]);
	}*/
	if(Varshost[0]+Varshost[1]==0.0) {
		for(i=0;i<npart;i++) {
			lhost[i]+=0.001;
			sigmahost[i]=(1.0/ALPHA)*lhost[i];
		}
	}
	else {
		for(i=0;i<npart;i++) {
			lhost[i]+=0.0001;
			sigmahost[i]=(1.0/ALPHA)*lhost[i];
		}
	}
	HANDLE_ERROR(hipMemcpy(l,lhost,npart*sizeof(double),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(sigma,sigmahost,npart*sizeof(double),hipMemcpyHostToDevice));
}

int main(int argc, char *argv[]) {
	if(argc>1) {
		doOpen(argv[1],"w");
	}
	else {
		fprintf(stderr,"No File Name Given\n");
	}
	const char header[4]="SP2";

	unsigned int n=npart;
	double alpha=ALPHA;
	double radius=R;
	double height=H;

	doWrite(header,4,1);
	doWrite(&n,sizeof(unsigned int),1);
	doWrite(&alpha,sizeof(double),1);
	doWrite(&radius,sizeof(double),1);
	doWrite(&height,sizeof(double),1);

	clock_t t2, t3;
	t2=clock();
	start();
	HANDLE_ERROR(hipMemcpy(sigmahost,sigma,npart*sizeof(double),hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(lhost,l,npart*sizeof(double),hipMemcpyDeviceToHost));
	int i;
	clock_t t0, t1;
	while(packfrac()<PHI) {
		t0=clock();
		doAnIter();
		t1=clock();
		int q;
		for(q=0;q<npart;q++) {
			doWrite((sigmahost+q),sizeof(double),1);
			doWrite((lhost+q),sizeof(double),1);
			doWrite((rhost+q),sizeof(double3),1);
			doWrite((uhost+q),sizeof(double3),1);
		}
		fprintf(stderr,"Iteration Time: %lf\n",(t1-t0)/((double)CLOCKS_PER_SEC));
	}
	for(i=0;i<npart;i++) {
		fprintf(stdout,"%lf	%lf	%lf	%lf	%lf	%lf\n",rhost[i].x,rhost[i].y,rhost[i].z,uhost[i].x,uhost[i].y,uhost[i].z);
	}
	t3=clock();
	fprintf(stderr,"Number of Particles: %i\n",npart);
	fprintf(stderr,"Total Time Taken: %lf\n",(t3-t2)/((double)CLOCKS_PER_SEC));
	doClose();
	return 0;
}
